#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "CImg.h"
#include <stdio.h>
#include <cstdlib>
#include <time.h>
#include<iostream>


#define BLOCK_SIZE 32
#define WA 512  
#define HA 512     
#define HC 3     
#define WC 3
#define WB (WA - WC + 1)
#define HB (HA - HC + 1)

using namespace std;
using namespace cimg_library;

// install cimg library
//$ sudo apt install cimg-dev
// compile using nvcc and -lX11 flag
//$ nvcc main.cu -lX11

__global__ void imageFilteringKernel( unsigned char *d_f, int paddedW, int paddedH,int *d_g, int S,unsigned char *d_h,int W,int H )
{

    // Set the padding size and filter size
    unsigned int paddingSize = S;
    unsigned int filterSize = 2 * S + 1;

    // Set the pixel coordinate
    const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;
    const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;

    // The multiply-add operation for the pixel coordinate ( j, i )
    if( j >= paddingSize && j < paddedW - paddingSize && i >= paddingSize && i < paddedH - paddingSize ) {
        unsigned int oPixelPos = ( i - paddingSize ) * W + ( j - paddingSize );
        d_h[oPixelPos] = 0.0;
        for( int k = -S; k <= S; k++ ) {
            for( int l = -S; l <= S; l++ ) {
                unsigned int iPixelPos = (( i + k ) * paddedW + ( j + l ));
                unsigned int coefPos = ( k + S ) * filterSize + ( l + S );
                
                d_h[oPixelPos] += d_f[iPixelPos] * d_g[coefPos];
            }
        }
    }

}

__global__ void rgb2gray(unsigned char * d_src, unsigned char * d_dst, int width, int height)
{
    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pos_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (pos_x >= width || pos_y >= height)
        return;

    /*
     * CImg RGB channels are split, not interleaved.
     * (http://cimg.eu/reference/group__cimg__storage.html)
     */
    unsigned char r = d_src[pos_y * width + pos_x];
    unsigned char g = d_src[(height + pos_y ) * width + pos_x];
    unsigned char b = d_src[(height * 2 + pos_y) * width + pos_x];

    unsigned int _gray = (unsigned int)((float)(r + g + b) / 3.0f + 0.5);
    unsigned char gray = _gray > 255 ? 255 : _gray;

    d_dst[pos_y * width + pos_x] = gray;
}


int main()
{
    //load image
    CImg<unsigned char> src("a.png");
    int width = src.width();
    int height = src.height();
    unsigned long size = src.size();
	cout<<size<<"--------------"<<endl;
    //create pointer to image
    unsigned char *h_src = src.data();
    int *h_filtro;

    CImg<unsigned char> dst(width, height, 1, 1);
    unsigned char *h_dst = dst.data();

    
    
    unsigned int size_C = HC * WC;
	unsigned int mem_size_C = sizeof(float) * size_C;

	unsigned char *d_src;
    unsigned char *d_dst;
    unsigned char *d_dst2;
	int *filtro = (int*)malloc(mem_size_C);
	for(int x=0;x<9;x++){
		filtro[0]=0;
	}

    hipMalloc((void**)&d_src, size);
    hipMalloc((void**)&d_dst, width*height*sizeof(unsigned char));
    hipMalloc((void**)&d_dst2, width*height*sizeof(unsigned char));

    hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice);
    hipMemcpy(h_filtro, filtro, size_C, hipMemcpyHostToDevice);



	int hFilterSize = 3 / 2;
    int paddedIWidth = width + 2 * hFilterSize;
    int paddedIHeight = height + 2 * hFilterSize;
    	
	
	
	
    //launch the kernel
    dim3 blkDim (16, 16, 1);
    dim3 grdDim ((width + 15)/16, (height + 15)/16, 1);
    
	/*rgb2gray<<<grdDim, blkDim>>>(d_src, d_dst, width, height);
    //wait until kernel finishes
    hipDeviceSynchronize();

    //copy back the result to CPU
    hipMemcpy(h_dst, d_dst, width*height, hipMemcpyDeviceToHost);*/
    
	
	
	imageFilteringKernel<<<grdDim,blkDim>>>( h_dst, paddedIWidth, paddedIHeight,
						h_filtro, hFilterSize,
						d_dst2, width, height );
						
	hipDeviceSynchronize();

	
    hipMemcpy(h_dst, d_dst2, width*height, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i){
	cout<<"i = "<<i<<" "<<(h_dst[i]*255)/255.0<<endl;

	}
						
    hipFree(d_src);
    hipFree(d_dst);
	
    CImgDisplay main_disp(dst, "After Processing");
    while (!main_disp.is_closed())
        main_disp.wait();

    return 0;
}
